#include "hip/hip_runtime.h"
﻿// Copyright (c) 2023 The InterpretML Contributors
// Licensed under the MIT license.
// Author: Paul Koch <code@koch.ninja>

#include "hip/hip_runtime.h"
#include ""

#include <type_traits>

#include "libebm.h"
#include "logging.h"
#include "common_c.h"
#include "bridge_c.h"
#include "zones.h"

#include "common_cpp.hpp"
#include "bridge_cpp.hpp"

#include "Registration.hpp"
#include "Objective.hpp"

#include "approximate_math.hpp"
#include "compute_stats.hpp"

namespace DEFINED_ZONE_NAME {
#ifndef DEFINED_ZONE_NAME
#error DEFINED_ZONE_NAME must be defined
#endif // DEFINED_ZONE_NAME

template <typename TObjective>
GPU_GLOBAL void TestGpuAdd(const Objective * const pObjective, const int * const pVal1, const int * const pVal2, int * const pResult) {
   TObjective * const pObjectiveSpecific = static_cast<TObjective *>(pObjective);
   const size_t iGpuThread = threadIdx.x;
//   pResult[iGpuThread] = static_cast<int>(static_cast<float>(pObjectiveSpecific->CalculateGradient(static_cast<float>(pVal1[iGpuThread]), static_cast<float>(pVal2[iGpuThread]))));
}

struct Cuda_32_Int final {
   static constexpr int cPack = 1;
   using T = uint32_t;

   GPU_BOTH inline Cuda_32_Int(const uint32_t val) noexcept : m_data(val) {
      UNUSED(m_data);
   }

private:
   uint32_t m_data;
};
static_assert(std::is_standard_layout<Cuda_32_Int>::value && std::is_trivially_copyable<Cuda_32_Int>::value,
   "This allows offsetof, memcpy, memset, inter-language, GPU and cross-machine use where needed");


struct Cuda_32_Float final {
   // https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__SINGLE.html#group__CUDA__MATH__SINGLE
   // https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__DOUBLE.html#group__CUDA__MATH__DOUBLE

   static constexpr bool bCpu = false;
   static constexpr int cPack = 1;
   using T = float;
   using TInt = Cuda_32_Int;

   WARNING_PUSH
   ATTRIBUTE_WARNING_DISABLE_UNINITIALIZED_MEMBER
   GPU_BOTH inline Cuda_32_Float() noexcept {
   }
   WARNING_POP

   Cuda_32_Float(const Cuda_32_Float & other) noexcept = default; // preserve POD status
   Cuda_32_Float & operator=(const Cuda_32_Float &) noexcept = default; // preserve POD status

   GPU_BOTH inline Cuda_32_Float(const double val) noexcept : m_data { static_cast<T>(val) } {
   }
   GPU_BOTH inline Cuda_32_Float(const float val) noexcept : m_data { static_cast<T>(val) } {
   }
   GPU_BOTH inline Cuda_32_Float(const int val) noexcept : m_data { static_cast<T>(val) } {
   }

   GPU_BOTH inline Cuda_32_Float & operator= (const double val) noexcept {
      m_data = static_cast<T>(val);
      return *this;
   }
   GPU_BOTH inline Cuda_32_Float & operator= (const float val) noexcept {
      m_data = static_cast<T>(val);
      return *this;
   }
   GPU_BOTH inline Cuda_32_Float & operator= (const int val) noexcept {
      m_data = static_cast<T>(val);
      return *this;
   }


   GPU_BOTH inline Cuda_32_Float operator+() const noexcept {
      return *this;
   }

   GPU_BOTH inline Cuda_32_Float operator-() const noexcept {
      return Cuda_32_Float(-m_data);
   }


   GPU_BOTH inline Cuda_32_Float operator+ (const Cuda_32_Float & other) const noexcept {
      return Cuda_32_Float(m_data + other.m_data);
   }

   GPU_BOTH inline Cuda_32_Float operator- (const Cuda_32_Float & other) const noexcept {
      return Cuda_32_Float(m_data - other.m_data);
   }

   GPU_BOTH inline Cuda_32_Float operator* (const Cuda_32_Float & other) const noexcept {
      return Cuda_32_Float(m_data * other.m_data);
   }

   GPU_BOTH inline Cuda_32_Float operator/ (const Cuda_32_Float & other) const noexcept {
      return Cuda_32_Float(m_data / other.m_data);
   }

   GPU_BOTH inline Cuda_32_Float & operator+= (const Cuda_32_Float & other) noexcept {
      *this = (*this) + other;
      return *this;
   }

   GPU_BOTH inline Cuda_32_Float & operator-= (const Cuda_32_Float & other) noexcept {
      *this = (*this) - other;
      return *this;
   }

   GPU_BOTH inline Cuda_32_Float & operator*= (const Cuda_32_Float & other) noexcept {
      *this = (*this) * other;
      return *this;
   }

   GPU_BOTH inline Cuda_32_Float & operator/= (const Cuda_32_Float & other) noexcept {
      *this = (*this) / other;
      return *this;
   }


   GPU_BOTH friend inline Cuda_32_Float operator+ (const double val, const Cuda_32_Float & other) noexcept {
      return Cuda_32_Float(val) + other;
   }

   GPU_BOTH friend inline Cuda_32_Float operator- (const double val, const Cuda_32_Float & other) noexcept {
      return Cuda_32_Float(val) - other;
   }

   GPU_BOTH friend inline Cuda_32_Float operator* (const double val, const Cuda_32_Float & other) noexcept {
      return Cuda_32_Float(val) * other;
   }

   GPU_BOTH friend inline Cuda_32_Float operator/ (const double val, const Cuda_32_Float & other) noexcept {
      return Cuda_32_Float(val) / other;
   }

   GPU_BOTH inline void LoadAligned(const T * const a) noexcept {
      m_data = *a;
   }

   GPU_BOTH inline void SaveAligned(T * const a) const noexcept {
      *a = m_data;
   }

   template<typename TFunc>
   GPU_BOTH friend inline Cuda_32_Float ApplyFunction(const Cuda_32_Float & val, const TFunc & func) noexcept {
      // this function is more useful for a SIMD operator where it applies func() to all packed items
      return Cuda_32_Float(func(val.m_data));
   }

   GPU_BOTH friend inline Cuda_32_Float IfGreater(const Cuda_32_Float & cmp1, const Cuda_32_Float & cmp2, const Cuda_32_Float & trueVal, const Cuda_32_Float & falseVal) noexcept {
      return cmp1.m_data > cmp2.m_data ? trueVal : falseVal;
   }

   GPU_BOTH friend inline Cuda_32_Float IfLess(const Cuda_32_Float & cmp1, const Cuda_32_Float & cmp2, const Cuda_32_Float & trueVal, const Cuda_32_Float & falseVal) noexcept {
      return cmp1.m_data < cmp2.m_data ? trueVal : falseVal;
   }

   GPU_BOTH friend inline Cuda_32_Float Sqrt(const Cuda_32_Float & val) noexcept {
      return Cuda_32_Float(sqrtf(val.m_data));
   }

   GPU_BOTH friend inline Cuda_32_Float Exp(const Cuda_32_Float & val) noexcept {
      return Cuda_32_Float(expf(val.m_data));
   }

   GPU_BOTH friend inline Cuda_32_Float Log(const Cuda_32_Float & val) noexcept {
      return Cuda_32_Float(logf(val.m_data));
   }

   GPU_BOTH friend inline T Sum(const Cuda_32_Float & val) noexcept {
      return val.m_data;
   }


   template<typename TObjective, size_t cCompilerScores, ptrdiff_t cCompilerPack, bool bHessian, bool bKeepGradHess, bool bCalcMetric, bool bWeight>
   INLINE_RELEASE_TEMPLATED static ErrorEbm OperatorApplyUpdate(const Objective * const pObjective, ApplyUpdateBridge * const pData) noexcept {
      static constexpr size_t k_cItems = 5;

      bool bExitError = true;

      const int aVal1[k_cItems] = { 5, 4, 3, 2, 1 };
      const int aVal2[k_cItems] = { 100, 200, 300, 400, 500 };
      int aResult[k_cItems];

      static_assert(std::is_standard_layout<TObjective>::value && std::is_trivially_copyable<TObjective>::value,
         "This allows offsetof, memcpy, memset, inter-language, GPU and cross-machine use where needed");

      int * aDeviceVal1 = nullptr;
      int * aDeviceVal2 = nullptr;
      int * aDeviceResult = nullptr;
      void * pDeviceObjective = nullptr;
      hipError_t error;

      error = hipSetDevice(0);
      if(hipSuccess != error) {
         goto exit_error;
      }

      error = hipMalloc((void **)&aDeviceVal1, k_cItems * sizeof(int));
      if(hipSuccess != error) {
         goto exit_error;
      }

      error = hipMalloc((void **)&aDeviceVal2, k_cItems * sizeof(int));
      if(hipSuccess != error) {
         goto exit_error;
      }

      error = hipMalloc((void **)&aDeviceResult, k_cItems * sizeof(int));
      if(hipSuccess != error) {
         goto exit_error;
      }

      if(!std::is_empty<TObjective>::value) {
         error = hipMalloc((void **)&pDeviceObjective, sizeof(TObjective));
         if(hipSuccess != error) {
            goto exit_error;
         }
         error = hipMemcpy(pDeviceObjective, pObjective, sizeof(TObjective), hipMemcpyHostToDevice);
         if(hipSuccess != error) {
            goto exit_error;
         }
      }

      error = hipMemcpy(aDeviceVal1, aVal1, k_cItems * sizeof(int), hipMemcpyHostToDevice);
      if(hipSuccess != error) {
         goto exit_error;
      }

      error = hipMemcpy(aDeviceVal2, aVal2, k_cItems * sizeof(int), hipMemcpyHostToDevice);
      if(hipSuccess != error) {
         goto exit_error;
      }

      TestGpuAdd<TObjective><<<1, k_cItems>>>(static_cast<Objective *>(pDeviceObjective), aDeviceVal1, aDeviceVal2, aDeviceResult);
      RemoteApplyUpdate<TObjective, cCompilerScores, cCompilerPack, bHessian, bKeepGradHess, bCalcMetric, bWeight><<<1, k_cItems>>>(pObjective, pData);

      error = hipGetLastError();
      if(hipSuccess != error) {
         goto exit_error;
      }

      error = hipDeviceSynchronize();
      if(hipSuccess != error) {
         goto exit_error;
      }

      error = hipMemcpy(aResult, aDeviceResult, k_cItems * sizeof(int), hipMemcpyDeviceToHost);
      if(hipSuccess != error) {
         goto exit_error;
      }

      bExitError = false;

   exit_error:

      bool bExitHard = false;

      if(nullptr != pDeviceObjective) {
         error = hipFree(pDeviceObjective);
         if(hipSuccess != error) {
            bExitHard = true;
         }
      }

      if(nullptr != aDeviceResult) {
         error = hipFree(aDeviceResult);
         if(hipSuccess != error) {
            bExitHard = true;
         }
      }

      if(nullptr != aDeviceVal2) {
         error = hipFree(aDeviceVal2);
         if(hipSuccess != error) {
            bExitHard = true;
         }
      }

      if(nullptr != aDeviceVal1) {
         error = hipFree(aDeviceVal1);
         if(hipSuccess != error) {
            bExitHard = true;
         }
      }

      if(bExitHard) {
         bExitError = true;

         // not much to do with the error if we fail hipDeviceReset after failing hipFree
         error = hipDeviceReset();
      }

      return bExitError ? Error_UnexpectedInternal : Error_None;
   }

private:

   float m_data;
};
static_assert(std::is_standard_layout<Cuda_32_Float>::value && std::is_trivially_copyable<Cuda_32_Float>::value,
   "This allows offsetof, memcpy, memset, inter-language, GPU and cross-machine use where needed");

// FIRST, define the RegisterObjective function that we'll be calling from our registrations.  This is a static 
// function, so we can have duplicate named functions in other files and they'll refer to different functions
template<template <typename> class TRegistrable, typename... Args>
INLINE_ALWAYS static std::shared_ptr<const Registration> RegisterObjective(const char * const sRegistrationName, const Args...args) {
   return Register<TRegistrable, Cuda_32_Float>(sRegistrationName, args...);
}

// now include all our special objective registrations which will use the RegisterObjective function we defined above!
#include "objective_registrations.hpp"

INTERNAL_IMPORT_EXPORT_BODY ErrorEbm CreateObjective_Cuda_32(
   const Config * const pConfig,
   const char * const sObjective,
   const char * const sObjectiveEnd,
   ObjectiveWrapper * const pObjectiveWrapperOut
) {
   return Objective::CreateObjective(&RegisterObjectives, pConfig, sObjective, sObjectiveEnd, pObjectiveWrapperOut);
}

} // DEFINED_ZONE_NAME
